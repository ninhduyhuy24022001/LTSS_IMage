    
#include <hip/hip_runtime.h>
#include <stdio.h>

    #define CHECK(call)\
    {\
        const hipError_t error = call;\
        if (error != hipSuccess)\
        {\
            fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);\
            fprintf(stderr, "code: %d, reason: %s\n", error,\
                    hipGetErrorString(error));\
            exit(EXIT_FAILURE);\
        }\
    }

    struct GpuTimer
    {
        hipEvent_t start;
        hipEvent_t stop;

        GpuTimer()
        {
            hipEventCreate(&start);
            hipEventCreate(&stop);
        }

        ~GpuTimer()
        {
            hipEventDestroy(start);
            hipEventDestroy(stop);
        }

        void Start()
        {
            hipEventRecord(start,0);
            hipEventSynchronize(start);
        }

        void Stop()
        {
            hipEventRecord(stop, 0);
        }

        float Elapsed()
        {
            float elapsed;
            hipEventSynchronize(stop);
            hipEventElapsedTime(&elapsed, start, stop);
            return elapsed;
        }
    };

    __global__ void addVecKernel(int *in1, int *in2, int n, 
            int *out)
    {
        int i = blockIdx.x * blockDim.x + threadIdx.x; 

        if (i < n)
        {
            out[i] = in1[i] + in2[i];
        }
    }

    void addVec(int *in1, int *in2, int n, 
            int *out, 
            bool useDevice=false, dim3 blockSize=dim3(1), int nStreams=1)
    {
        if (useDevice == false)
        {
            for (int i = 0; i < n; i++)
            {
                out[i] = in1[i] + in2[i];
            }
        }
        else // Use device
        {
            hipDeviceProp_t devProp;
            hipGetDeviceProperties(&devProp, 0);
            printf("GPU name: %s\n", devProp.name);
            printf("GPU compute capability: %d.%d\n", devProp.major, devProp.minor);
            
            // Pin host memory regions (allocated by malloc)
            // so that we can use cudaMemcpyAsync  
            size_t nBytes = n * sizeof(int);
            CHECK(hipHostRegister(in1, nBytes, hipHostRegisterDefault));
            CHECK(hipHostRegister(in2, nBytes, hipHostRegisterDefault));
            CHECK(hipHostRegister(out, nBytes, hipHostRegisterDefault));

            // TODO: Allocate device memory regions
            int *d_in1, *d_in2, *d_out;
            CHECK(hipHostMalloc(&d_in1, nBytes, hipHostMallocDefault));
            CHECK(hipHostMalloc(&d_in2, nBytes, hipHostMallocDefault));
            CHECK(hipMalloc(&d_out, nBytes))

            // TODO: Create "nStreams" device streams
            hipStream_t* streams = (hipStream_t*)malloc(nStreams * sizeof(hipStream_t));
            for (int i = 0; i < nStreams; i++){
                hipStreamCreate(&streams[i]);
            }
                // Create event
            hipEvent_t startEvent, stopEvent;
            CHECK(hipEventCreate(&startEvent));
            CHECK(hipEventCreate(&stopEvent));
                // Send start event to null stream;

            GpuTimer timer;
            timer.Start();

            // TODO: Send jobs (H2D, kernel, D2H) to device streams
            int streamSize = n/nStreams + 1;
            // int streamBytes = streamSize*sizeof(int);
            // dim3 gridSize((streamSize - 1) / blockSize.x + 1);


            hipEventRecord(startEvent, 0);
            for (int i = 0; i < nStreams; i++){
                int offset;
                int offsetIndex = i*streamSize;
                if (i == (nStreams-1)){
                    offset = (n - i*streamSize);
                }
                else
                    offset =  streamSize;

                dim3 gridSize((offset - 1) / blockSize.x + 1);
                int offsetBytes = offset*sizeof(int);

                // printf("%d\n", offsetIndex);

                CHECK(hipMemcpyAsync(&d_in1[offsetIndex], &in1[offsetIndex], 
                                    offsetBytes, hipMemcpyHostToDevice, streams[i]));
                CHECK(hipMemcpyAsync(&d_in2[offsetIndex], &in2[offsetIndex], 
                                    offsetBytes, hipMemcpyHostToDevice, streams[i]));
                addVecKernel<<<gridSize, blockSize, 0,  streams[i]>>>(&d_in1[offsetIndex], &d_in2[offsetIndex], offset, &d_out[offsetIndex]);
                CHECK(hipMemcpyAsync(&out[offsetIndex], &d_out[offsetIndex],
                                    offsetBytes, hipMemcpyDeviceToHost, streams[i]));
            }


            CHECK(hipEventRecord(stopEvent, 0));
            CHECK(hipEventSynchronize(stopEvent));

            timer.Stop();
            float time = timer.Elapsed();
            printf("Processing time of all device streams: %f ms\n\n", time);

            // TODO: Destroy device streams
            CHECK(hipEventDestroy(startEvent));
            CHECK(hipEventDestroy(stopEvent));

            for (int i = 0; i < nStreams; i++){
                hipStreamDestroy(streams[i]);
            }

            // TODO: Free device memory regions
            hipHostFree(d_out);
            hipHostFree(d_in1);
            hipHostFree(d_in2);

            // Unpin host memory regions
            CHECK(hipHostUnregister(in1));
            CHECK(hipHostUnregister(in2));
            CHECK(hipHostUnregister(out));
        }
    }

    int main(int argc, char ** argv)
    {
        int n; 
        int *in1, *in2; 
        int *out, *correctOut;

        // Input data into n
        n = (1 << 24) + 1;
        printf("n =  %d\n\n", n);

        // Allocate memories for in1, in2, out
        size_t nBytes = n * sizeof(int);
        in1 = (int *)malloc(nBytes);
        in2 = (int *)malloc(nBytes);
        out = (int *)malloc(nBytes);
        /*
        CHECK(cudaMallocHost(&in1, nBytes));
        CHECK(cudaMallocHost(&in2, nBytes));
        CHECK(cudaMallocHost(&out, nBytes));
        */
        correctOut = (int *)malloc(nBytes);

        // Input data into in1, in2
        for (int i = 0; i < n; i++)
        {
            in1[i] = rand() & 0xff; // Random int in [0, 255]
            in2[i] = rand() & 0xff; // Random int in [0, 255]
        }

        // Add in1 & in2 on host
        addVec(in1, in2, n, correctOut);

        // Add in1 & in2 on device
        dim3 blockSize(512); // Default
        int nStreams = 1; // Default
        if (argc >= 2)
        {
            blockSize.x = atoi(argv[1]);
            if (argc >= 3)
            {
                nStreams = atoi(argv[2]);
            }
        } 
        addVec(in1, in2, n, out, true, blockSize, nStreams);

        for (int i = 0; i < n; i++)
        {
            if (out[i] != correctOut[i])
            {
                printf("INCORRECT :(\n");
                return 1;
            }
        }

        printf("CORRECT :)\n");

        free(in1);
        free(in2);
        free(out);
        /*
        CHECK(cudaFreeHost(in1));
        CHECK(cudaFreeHost(in2));
        CHECK(cudaFreeHost(out));
        */
        free(correctOut);
    }
